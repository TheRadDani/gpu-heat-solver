#include "hip/hip_runtime.h"
#include <heat-equations.h>

__global__ void jacobi_iteration(float* temperature_old, float* temperature_new)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= 1 && row < HEIGHT - 1 && col >= 1 && col < WIDTH - 1) {
        int index = row * WIDTH + col;
        temperature_new[index] = (temperature_old[index - 1] + temperature_old[index + 1] +
                                  temperature_old[index - WIDTH] + temperature_old[index + WIDTH]) / 4.0f;
    }
}

int main()
{
    float* temperature_old, * temperature_new;
    hipMallocManaged(&temperature_old, WIDTH * HEIGHT * sizeof(float));
    hipMallocManaged(&temperature_new, WIDTH * HEIGHT * sizeof(float));

    // Initialize temperature_old to a constant value
    for (int i = 0; i < WIDTH * HEIGHT; i++) {
        temperature_old[i] = 25.0f;
    }

    // Set boundary conditions (fixed temperature on all four sides)
    for (int i = 0; i < WIDTH; i++) {
        temperature_old[i] = 100.0f;
        temperature_old[(HEIGHT - 1) * WIDTH + i] = 0.0f;
    }
    for (int i = 0; i < HEIGHT; i++) {
        temperature_old[i * WIDTH] = 75.0f;
        temperature_old[(i + 1) * WIDTH - 1] = 50.0f;
    }

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_blocks(WIDTH / BLOCK_SIZE, HEIGHT / BLOCK_SIZE);

    for (int i = 0; i < NUM_ITERATIONS; i++) {
        jacobi_iteration<<<num_blocks, block_size>>>(temperature_old, temperature_new);
        hipDeviceSynchronize();

        // Swap old and new temperature arrays
        float* temp = temperature_old;
        temperature_old = temperature_new;
        temperature_new = temp;
    }

    // Print out the final temperature values
    for (int row = 0; row < HEIGHT; row++) {
        for (int col = 0; col < WIDTH; col++) {
            printf("%.2f ", temperature_old[row * WIDTH + col]);
        }
        printf("\n");
    }

    hipFree(temperature_old);
    hipFree(temperature_new);

    return 0;
}
